#include <stdio.h>
#include <hip/hip_runtime.h>

void cudaCheckError(hipError_t error, const char* file, const int line) {
  if (error != hipSuccess) {
    printf("CUDA error (%s:%d): %s\n", file, line, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define CUDA_CHECK_ERROR(error) cudaCheckError(error, __FILE__, __LINE__)

void hipLaunchKernel(void (*func)(void*), const void* args, const int blockSize, const int numBlocks) {
  func<<<numBlocks, blockSize>>>(args);
  CUDA_CHECK_ERROR(hipPeekAtLastError());
  CUDA_CHECK_ERROR(hipDeviceSynchronize());
}
